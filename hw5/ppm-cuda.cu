// nvcc -Xcompiler -Wall -DDOLOG ppm-cuda.cu


#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>
#include <cstdio>

// https://stackoverflow.com/questions/28896001/read-write-to-ppm-image-file-c

#ifdef DOLOG
#define LOG(msg) std::cerr<<msg<<std::endl
//#define LOG(msg) fprintf(stderr, msg "\n");
#else
#define LOG(msg)
#endif

// host code for validating last cuda operation (not kernel launch)

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



char* data;

int read(std::string filename,
         int& width,
         int& height,
         std::vector<float>& r,
         std::vector<float>& g,
         std::vector<float>& b)
{
    std::ifstream in(filename.c_str(), std::ios::binary);

    int maxcol;

    if (! in.is_open())
    {
        std::cerr << "could not open " << filename << " for reading" << std::endl;
        return 0;
    }

    {
        std::string magicNum;
        in >> magicNum;
        LOG("got magicNum:" << magicNum);

        // this is broken if magicNum != 'P6'
    }

    {
        long loc = in.tellg();
        std::string comment;
        in >> comment;

        if (comment[0] != '#')
        {
            in.seekg(loc);
        }
        else
        {
            LOG("got comment:" << comment);
        }
    }

    in >> width >> height >> maxcol;
    in.get();                   // eat newline
    LOG("dimensions: " << width << "x" << height << "("<<maxcol<<")");
    

//    char* data = new char[width*height*3];
    data = new char[width*height*3];
    in.read(data, width*height*3);
    in.close();
    
    r.resize(width*height);
    g.resize(width*height);
    b.resize(width*height);

    for (int i=0; i<width*height; ++i)
    {
        int base = i*3;
        r[i] =  ((unsigned char)data[base+0])/255.0f;
        g[i] =  ((unsigned char)data[base+1])/255.0f;
        b[i] =  ((unsigned char)data[base+2])/255.0f;
    }
    free(data);

    return 1;
}


int write(std::string outfile,
          int width, int height,
          const std::vector<float>& r,
          const std::vector<float>& g,
          const std::vector<float>& b)
{
    std::ofstream ofs(outfile.c_str(), std::ios::out | std::ios::binary);

    if (! ofs.is_open())
    {
        std::cerr << "could not open " << outfile << " for writing" << std::endl;
    }

    ofs << "P6\n#*\n" << width << " " << height << "\n255\n";

    for (int i=0; i < width*height; ++i)
    {
        ofs <<
            (unsigned char)(r[i]*255) <<
            (unsigned char)(g[i]*255) <<
            (unsigned char)(b[i]*255);
    }
    ofs.close();
    
    return 1;
}



#define imin(a,b) (a<b?a:b)

__global__ void process(int width, int height, float* r, float* g, float* b)
{
    // thread's .x coordinates are the pixel column in the image
    // thread's .y coordinates are the pixel row in the image

    int global_pixel_row=threadIdx.y + blockIdx.y*blockDim.y;
    int global_pixel_col=threadIdx.x + blockIdx.x*blockDim.x;

    if (global_pixel_col < width &&  global_pixel_row < height)
    {
        // this pixel exists in the image and this is not an idle thread

        // find the index in r,g,b for this pixel
        int index=global_pixel_col + global_pixel_row * width;

        // draw thread block boundaries
        if (threadIdx.x==0 || threadIdx.x==blockDim.x-1 ||
            threadIdx.y==0 || threadIdx.y == blockDim.y-1)
        {
            r[index]=1;
            g[index]=0;
            b[index]=0;
        }
        else
        {
            r[index]=1;
            g[index]=1;
            b[index]=1;
        }
            
    }
    else
    {
        // do nothing
    }





    // image data (r,g,b) is stored row-major (all of pixel row 0, followed by all of pixel row 1, etc.)
    

    

}



int main(int argc, char *argv[])
{
    int width, height;

    std::vector<float> r,g,b;
    float *d_r, *d_g, *d_b;

#ifdef DO_READ

    read("input.ppm", width, height, r,g,b);
    LOG("processing " << width << "x" << height);


    // copy read image to GPU
    gpuErrchk(cudaMalloc(&d_r, width*height*sizeof(float)));
    gpuErrchk(cudaMalloc(&d_g, width*height*sizeof(float)));
    gpuErrchk(cudaMalloc(&d_b, width*height*sizeof(float)));

    gpuErrchk(cudaMemcpy(d_r, &r[0], width*height*sizeof(float), cudaMemcpyHostToDevice));
    gpuErrchk(cudaMemcpy(d_g, &g[0], width*height*sizeof(float), cudaMemcpyHostToDevice));
    gpuErrchk(cudaMemcpy(d_b, &b[0], width*height*sizeof(float), cudaMemcpyHostToDevice));
#else

    width=640;
    height=480;

    gpuErrchk(hipMalloc(&d_r, width*height*sizeof(float)));
    gpuErrchk(hipMalloc(&d_g, width*height*sizeof(float)));
    gpuErrchk(hipMalloc(&d_b, width*height*sizeof(float)));

#endif
    

    
    // call kernel

    dim3 tpb(50, 12);
    //dim3 tpb(1, 1);
    dim3 bpg((width+tpb.x-1)/tpb.x, (height+tpb.y-1)/tpb.y);

    // for example ....
    process<<<bpg,tpb>>>(width, height, d_r, d_g, d_b);
    // check to see if there were any issues with the previous kernel launch
    gpuErrchk( hipPeekAtLastError() );



    // copy data back from kernel
    r.resize(width*height);
    g.resize(width*height);
    b.resize(width*height);
    
    gpuErrchk(hipMemcpy(&r[0], d_r, width*height*sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(&g[0], d_g, width*height*sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(&b[0], d_b, width*height*sizeof(float), hipMemcpyDeviceToHost));

    hipFree(d_r);
    hipFree(d_g);
    hipFree(d_b);
   
    // save image
    write("output.ppm", width, height, r,g,b);

    return 0;
}